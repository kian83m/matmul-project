
#include <hip/hip_runtime.h>
// #include <cuda_runtime.h>
// #include <cuda.h>
// #include <stdio.h>
// #include <math.h>

// #define CEIL_DIV(a, b) ((a + b - 1) / b)

// // Tile sizes
// #define BM 64 // Block size in M dimension
// #define BN 64 // Block size in N dimension
// #define BK 16 // Block size in K dimension

// // Thread tile sizes (each thread computes a 4x4 tile of C)
// #define TM 4
// #define TN 4

// __global__ void square_dgemm_kernel_2d_blocktiling(const int M,
//                                                    const double *__restrict__ A,
//                                                    const double *__restrict__ B,
//                                                    double *__restrict__ C)
// {
//   __shared__ double As[BM * BK]; // Shared memory for A tile
//   __shared__ double Bs[BK * BN]; // Shared memory for B tile

//   const int blockRow = blockIdx.x; // Along M dimension
//   const int blockCol = blockIdx.y; // Along N dimension

//   const int THREADS_PER_BLOCK_M = BM / TM;
//   const int THREADS_PER_BLOCK_N = BN / TN;

//   const int tRow = threadIdx.y;
//   const int tCol = threadIdx.x;

//   const int globalRowStart = blockRow * BM;
//   const int globalColStart = blockCol * BN;

//   const int threadBaseRow = tRow * TM;
//   const int threadBaseCol = tCol * TN;

//   double threadResults[TM * TN] = {0.0};

//   // Vectorized loads: each double2 loads two doubles.
//   // Total threads per block:
//   int totalThreads = THREADS_PER_BLOCK_M * THREADS_PER_BLOCK_N;
//   int linearThreadIdx = tRow * THREADS_PER_BLOCK_N + tCol;

//   // For A (BM x BK):
//   // Total elements = BM*BK. double2 load covers 2 elements.
//   int aVectorLoads = (BM * BK) / 2 / totalThreads;
//   // For B (BK x BN):
//   int bVectorLoads = (BK * BN) / 2 / totalThreads;

//   for (int kBlock = 0; kBlock < M; kBlock += BK)
//   {
//     // We define tile pointers for convenience
//     const double *A_tile = &A[globalRowStart * M + kBlock];
//     const double *B_tile = &B[kBlock * M + globalColStart];

//     {
//       double2 *AsVec = (double2 *)As;
//       for (int i = 0; i < aVectorLoads; i++)
//       {
//         int vecIndex = linearThreadIdx + i * totalThreads;
//         int elemIndex = vecIndex * 2;
//         if (elemIndex < BM * BK)
//         {
//           int rowA = elemIndex / BK;
//           int colA = elemIndex % BK;
//           int globalArow = globalRowStart + rowA;
//           int globalAcol = kBlock + colA;

//           double val0 = 0.0, val1 = 0.0;
//           // Check if both elements fit in range
//           if (globalArow < M && globalAcol < M)
//             val0 = A[globalArow * M + globalAcol];

//           // Second element of the pair
//           int nextColA = globalAcol + 1;
//           if ((colA + 1 < BK) && globalArow < M && nextColA < M)
//             val1 = A[globalArow * M + nextColA];

//           AsVec[vecIndex] = make_double2(val0, val1);
//         }
//       }
//     }

//     {
//       double2 *BsVec = (double2 *)Bs;
//       for (int i = 0; i < bVectorLoads; i++)
//       {
//         int vecIndex = linearThreadIdx + i * totalThreads;
//         int elemIndex = vecIndex * 2;
//         if (elemIndex < BK * BN)
//         {
//           int rowB = elemIndex / BN;
//           int colB = elemIndex % BN;
//           int globalBrow = kBlock + rowB;
//           int globalBcol = globalColStart + colB;

//           double val0 = 0.0, val1 = 0.0;
//           if (globalBrow < M && globalBcol < M)
//             val0 = B[globalBrow * M + globalBcol];

//           int nextColB = globalBcol + 1;
//           if ((colB + 1 < BN) && globalBrow < M && nextColB < M)
//             val1 = B[globalBrow * M + nextColB];

//           BsVec[vecIndex] = make_double2(val0, val1);
//         }
//       }
//     }

//     __syncthreads();

//     // Compute partial results for this K-block
//     for (int dotIdx = 0; dotIdx < BK; ++dotIdx)
//     {
//       double regM[TM];
//       double regN[TN];

//       for (int i = 0; i < TM; i++)
//       {
//         int aRow = threadBaseRow + i;
//         regM[i] = As[aRow * BK + dotIdx];
//       }

//       for (int j = 0; j < TN; j++)
//       {
//         int bCol = threadBaseCol + j;
//         regN[j] = Bs[dotIdx * BN + bCol];
//       }

//       for (int i = 0; i < TM; i++)
//       {
//         for (int j = 0; j < TN; j++)
//         {
//           threadResults[i * TN + j] += regM[i] * regN[j];
//         }
//       }
//     }

//     __syncthreads();
//   }

//   // Store results to C
//   for (int i = 0; i < TM; i++)
//   {
//     int cRow = globalRowStart + threadBaseRow + i;
//     for (int j = 0; j < TN; j++)
//     {
//       int cCol = globalColStart + threadBaseCol + j;
//       if (cRow < M && cCol < M)
//       {
//         C[cRow * M + cCol] = threadResults[i * TN + j];
//       }
//     }
//   }
// }

// double *dA, *dB, *dC;

// void square_dgemm(const int M, const double *A0, const double *B0, double *C0)
// {
//   cudaMalloc(&dA, M * M * sizeof(double));
//   cudaMalloc(&dB, M * M * sizeof(double));
//   cudaMalloc(&dC, M * M * sizeof(double));
//   cudaMemcpy(dA, A0, M * M * sizeof(double), cudaMemcpyHostToDevice);
//   cudaMemcpy(dB, B0, M * M * sizeof(double), cudaMemcpyHostToDevice);

//   dim3 numBlocks(CEIL_DIV(M, BM), CEIL_DIV(M, BN));
//   dim3 threadsPerBlock(BN / TN, BM / TM);

//   square_dgemm_kernel_2d_blocktiling<<<numBlocks, threadsPerBlock>>>(M, dA, dB, dC);
//   cudaDeviceSynchronize();
//   cudaMemcpy(C0, dC, M * M * sizeof(double), cudaMemcpyDeviceToHost);

//   cudaFree(dA);
//   cudaFree(dB);
//   cudaFree(dC);
// }

// const char *dgemm_desc = "Optimized DGEMM with partial vectorization (Step-by-step fix)";

// 2

// #include <cuda_runtime.h>
// #include <cuda.h>
// #include <stdio.h>
// #include <math.h>

// #define CEIL_DIV(a, b) ((a + b - 1) / b)

// // Tile sizes
// #define BM 64
// #define BN 64
// #define BK 16

// // Each thread computes a 4x4 tile
// #define TM 4
// #define TN 4

// __global__ void square_dgemm_kernel_2d_blocktiling(const int M,
//                                                    const double *__restrict__ A,
//                                                    const double *__restrict__ B,
//                                                    double *__restrict__ C)
// {
//   __shared__ double As[BM * BK]; // Shared memory for A tile
//   __shared__ double Bs[BK * BN]; // Shared memory for B tile

//   const int blockRow = blockIdx.x;
//   const int blockCol = blockIdx.y;

//   // Threads per block in M and N directions
//   const int THREADS_PER_BLOCK_M = BM / TM; // 64/4 =16
//   const int THREADS_PER_BLOCK_N = BN / TN; // 64/4 =16

//   const int tRow = threadIdx.y; // 0..15
//   const int tCol = threadIdx.x; // 0..15

//   const int globalRowStart = blockRow * BM;
//   const int globalColStart = blockCol * BN;

//   const int threadBaseRow = tRow * TM;
//   const int threadBaseCol = tCol * TN;

//   double threadResults[TM * TN] = {0.0};

//   const int totalThreads = THREADS_PER_BLOCK_M * THREADS_PER_BLOCK_N; // 256
//   const int linearThreadIdx = tRow * THREADS_PER_BLOCK_N + tCol;

//   // Number of double2 loads per thread for A and B
//   const int aVectorLoads = ((BM * BK) / 2 + totalThreads - 1) / totalThreads;
//   const int bVectorLoads = ((BK * BN) / 2 + totalThreads - 1) / totalThreads;

//   // Iterate over the K dimension in steps of BK
//   for (int kBlock = 0; kBlock < M; kBlock += BK)
//   {
//     // Pointers to the sub-block in global memory
//     const double *A_tile = A + (globalRowStart * M + kBlock);
//     const double *B_tile = B + (kBlock * M + globalColStart);

//     // Load A tile into shared memory using double2
//     for (int i = 0; i < aVectorLoads; i++)
//     {
//       int vecIndex = linearThreadIdx + i * totalThreads;
//       if (vecIndex < (BM * BK) / 2)
//       {
//         int elemIndex = vecIndex * 2;
//         int rowA = elemIndex / BK; // which row in A tile
//         int colA = elemIndex % BK; // which column in A tile

//         // First element
//         double val0 = 0.0;
//         int gRowA = globalRowStart + rowA;
//         int gColA = kBlock + colA;
//         if (gRowA < M && gColA < M)
//           val0 = A_tile[rowA * M + colA];

//         // Second element
//         double val1 = 0.0;
//         if (colA + 1 < BK)
//         {
//           int gColA2 = kBlock + (colA + 1);
//           if (gRowA < M && gColA2 < M)
//             val1 = A_tile[rowA * M + (colA + 1)];
//         }

//         As[elemIndex] = val0;
//         As[elemIndex + 1] = val1;
//       }
//     }

//     // Load B tile into shared memory using double2
//     for (int i = 0; i < bVectorLoads; i++)
//     {
//       int vecIndex = linearThreadIdx + i * totalThreads;
//       if (vecIndex < (BK * BN) / 2)
//       {
//         int elemIndex = vecIndex * 2;
//         int rowB = elemIndex / BN; // which row in B tile
//         int colB = elemIndex % BN; // which column in B tile

//         double val0 = 0.0;
//         int gRowB = kBlock + rowB;
//         int gColB = globalColStart + colB;
//         if (gRowB < M && gColB < M)
//           val0 = B_tile[rowB * M + colB];

//         double val1 = 0.0;
//         if (colB + 1 < BN)
//         {
//           int gColB2 = globalColStart + (colB + 1);
//           if (gRowB < M && gColB2 < M)
//             val1 = B_tile[rowB * M + (colB + 1)];
//         }

//         Bs[elemIndex] = val0;
//         Bs[elemIndex + 1] = val1;
//       }
//     }

//     __syncthreads();

//     // Compute partial results for this K-block
//     // dotIdx: which column in A and row in B
//     for (int dotIdx = 0; dotIdx < BK; ++dotIdx)
//     {
//       for (int i = 0; i < TM; i++)
//       {
//         double aVal = As[(threadBaseRow + i) * BK + dotIdx];
//         for (int j = 0; j < TN; j++)
//         {
//           double bVal = Bs[dotIdx * BN + (threadBaseCol + j)];
//           threadResults[i * TN + j] += aVal * bVal;
//         }
//       }
//     }

//     __syncthreads();
//   }

//   // Store results to global memory (scalar store for correctness)
//   for (int i = 0; i < TM; i++)
//   {
//     int cRow = globalRowStart + threadBaseRow + i;
//     if (cRow < M)
//     {
//       for (int j = 0; j < TN; j++)
//       {
//         int cCol = globalColStart + threadBaseCol + j;
//         if (cCol < M)
//         {
//           C[cRow * M + cCol] = threadResults[i * TN + j];
//         }
//       }
//     }
//   }
// }

// double *dA, *dB, *dC;

// void square_dgemm(const int M, const double *A0, const double *B0, double *C0)
// {
//   cudaMalloc(&dA, M * M * sizeof(double));
//   cudaMalloc(&dB, M * M * sizeof(double));
//   cudaMalloc(&dC, M * M * sizeof(double));

//   cudaMemcpy(dA, A0, M * M * sizeof(double), cudaMemcpyHostToDevice);
//   cudaMemcpy(dB, B0, M * M * sizeof(double), cudaMemcpyHostToDevice);

//   dim3 numBlocks(CEIL_DIV(M, BM), CEIL_DIV(M, BN));
//   dim3 threadsPerBlock(BN / TN, BM / TM); // (64/4=16, 64/4=16) -> (16,16)

//   square_dgemm_kernel_2d_blocktiling<<<numBlocks, threadsPerBlock>>>(M, dA, dB, dC);
//   cudaDeviceSynchronize();

//   cudaMemcpy(C0, dC, M * M * sizeof(double), cudaMemcpyDeviceToHost);

//   cudaFree(dA);
//   cudaFree(dB);
//   cudaFree(dC);
// }

// const char *dgemm_desc = "Corrected DGEMM with vectorized global loads using double2 and scalar computation.";

// Good First Step

// #include <cuda_runtime.h>
// #include <cuda.h>
// #include <stdio.h>
// #include <math.h>

// #define CEIL_DIV(a, b) ((a + b - 1) / b)

// // Tile sizes
// #define BM 64 // Block size in M dimension
// #define BN 64 // Block size in N dimension
// #define BK 16 // Block size in K dimension

// // Thread tile sizes (each thread computes a TMxTN tile of C)
// #define TM 4
// #define TN 4

// __global__ void square_dgemm_kernel_2d_blocktiling(const int M, const double *__restrict__ A, const double *__restrict__ B, double *__restrict__ C)
// {
//   __shared__ double As[BM * BK]; // now viewed as (BK x BM) in col-major form: As[k*BM + m]
//   __shared__ double Bs[BK * BN]; // viewed as (BK x BN): Bs[k*BN + n]

//   const int blockRow = blockIdx.x; // block index along M dimension
//   const int blockCol = blockIdx.y; // block index along N dimension

//   // Threads per block configuration:
//   const int THREADS_PER_BLOCK_M = BM / TM; // = 64/4 = 16
//   const int THREADS_PER_BLOCK_N = BN / TN; // = 64/4 = 16

//   // Total threads per block = 16*16 = 256 threads
//   // (But original code used BM=64, BN=64, TM=8, TN=8 => 8x8=64 threads.
//   // Adjusting TM,TN as per the original code:
//   // The user code states TM=4, TN=4 => THREADS_PER_BLOCK_M=16, THREADS_PER_BLOCK_N=16 => 256 threads.)
//   // If you prefer fewer threads, adjust TM,TN accordingly.

//   const int tRow = threadIdx.y;
//   const int tCol = threadIdx.x;

//   // Global start indices for this block tile
//   const int globalRowStart = blockRow * BM;
//   const int globalColStart = blockCol * BN;

//   // This thread's starting tile offset
//   const int threadBaseRow = tRow * TM;
//   const int threadBaseCol = tCol * TN;

//   // Flatten thread index for linear mapping of loads
//   const int THREADS_PER_BLOCK = THREADS_PER_BLOCK_M * THREADS_PER_BLOCK_N; // = 256
//   const int linearThreadIdx = tRow * THREADS_PER_BLOCK_N + tCol;

//   // Precompute how many elements each thread loads from A and B per iteration
//   const int elemsA = BM * BK; // total elements in the A tile
//   const int elemsB = BK * BN; // total elements in the B tile

//   // Each thread loads a contiguous chunk of these
//   // Using double2 vectorization: each double2 covers 2 elements
//   // Make sure (elemsA and elemsB) are even. They are 64*16=1024 for both, which is even.
//   const int double2_elemsA = elemsA / 2;
//   const int double2_elemsB = elemsB / 2;

//   const int elemsPerThreadA = (double2_elemsA + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
//   const int elemsPerThreadB = (double2_elemsB + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

//   double threadResults[TM * TN] = {0.0};

//   for (int kBlock = 0; kBlock < M; kBlock += BK)
//   {
//     //==========================
//     // Load A tile into shared memory (transposed)
//     //==========================
//     {
//       double2 *A2 = (double2 *)A; // reinterpret for vector loads
//       double2 *As2 = (double2 *)As;
//       int A_tile_double2_offset = (globalRowStart * M + kBlock);
//       // Note: We are loading a BMxBK block from A starting at (globalRowStart,kBlock)
//       // Flattened: row major in global mem: For i in [0,BM), j in [0,BK):
//       // globalAindex = (globalRowStart + i)*M + (kBlock + j)
//       // We want to store transposed in SMEM: As[k*BM + i]
//       // We'll just load row-major from global and store to As in col-major form.

//       for (int load_i = 0; load_i < elemsPerThreadA; load_i++)
//       {
//         int idx2 = linearThreadIdx + load_i * THREADS_PER_BLOCK;
//         if (idx2 < double2_elemsA)
//         {
//           // double2 index back to element index:
//           int elem_idx = idx2 * 2;  // each double2 covers 2 elements
//           int irow = elem_idx / BK; // i in [0,BM)
//           int icol = elem_idx % BK; // j in [0,BK)
//           int globalArow = globalRowStart + irow;
//           int globalAcol = kBlock + icol;
//           double2 val2 = {0.0, 0.0};

//           if (globalArow < M && globalAcol + 1 < M)
//           {
//             // load two elements at once
//             int globalAIdx2 = (globalArow * M + globalAcol) / 2;
//             // This doesn't match directly if we do double2 indexing simply.
//             // Let's load element-wise instead for clarity:
//             // We'll just do two single loads for clarity. Vectorization can be done if pointers are aligned.
//             double val0 = (globalArow < M && globalAcol < M) ? A[globalArow * M + globalAcol] : 0.0;
//             double val1 = (globalArow < M && (globalAcol + 1) < M) ? A[globalArow * M + globalAcol + 1] : 0.0;
//             val2.x = val0;
//             val2.y = val1;
//           }
//           else
//           {
//             // Handle boundary
//             if (globalArow < M && globalAcol < M)
//             {
//               val2.x = A[globalArow * M + globalAcol];
//             }
//             // val2.y remains 0 if out of range
//           }

//           // Now we must store them transposed in As:
//           // We have two consecutive elements: (irow, icol) and possibly (irow, icol+1)
//           // Actually, we must be careful here because we took two consecutive global elements.
//           // We are vector-loading horizontally along K dimension. It's simpler to consider we load along row:
//           // To simplify, let's just store element by element:
//           // We'll store val2.x and val2.y separately:
//           As[icol * BM + irow] = val2.x; // transpose: (irow,icol)->(icol,irow)
//           if (icol + 1 < BK)
//           {
//             As[(icol + 1) * BM + irow] = val2.y;
//           }
//         }
//       }
//     }

//     //==========================
//     // Load B tile into shared memory
//     //==========================
//     {
//       double2 *B2 = (double2 *)B;
//       double2 *Bs2 = (double2 *)Bs;
//       // Similar approach for B
//       // B tile: size BKxBN at (kBlock, globalColStart)
//       // globalBindex = (kBlock+i)*M + (globalColStart + j)
//       // We'll store similarly as Bs[k*BN + n]

//       for (int load_i = 0; load_i < elemsPerThreadB; load_i++)
//       {
//         int idx2 = linearThreadIdx + load_i * THREADS_PER_BLOCK;
//         if (idx2 < double2_elemsB)
//         {
//           int elem_idx = idx2 * 2;
//           int irow = elem_idx / BN; // i in [0,BK)
//           int icol = elem_idx % BN; // j in [0,BN)
//           int globalBrow = kBlock + irow;
//           int globalBcol = globalColStart + icol;
//           double val0 = 0.0, val1 = 0.0;
//           if (globalBrow < M && globalBcol < M)
//             val0 = B[globalBrow * M + globalBcol];
//           if (globalBrow < M && (globalBcol + 1) < M)
//             val1 = B[globalBrow * M + globalBcol + 1];

//           Bs[irow * BN + icol] = val0;
//           if (icol + 1 < BN)
//           {
//             Bs[irow * BN + (icol + 1)] = val1;
//           }
//         }
//       }
//     }

//     __syncthreads();

//     //==========================
//     // Compute partial results for this block
//     //==========================
//     for (int dotIdx = 0; dotIdx < BK; ++dotIdx)
//     {
//       double regM[TM];
//       double regN[TN];
//       for (int i = 0; i < TM; i++)
//       {
//         int aRow = threadBaseRow + i;
//         // As is transposed: As[k*BM + m], with k=dotIdx, m=aRow
//         regM[i] = As[dotIdx * BM + aRow];
//       }
//       for (int j = 0; j < TN; j++)
//       {
//         int bCol = threadBaseCol + j;
//         // Bs[k*BN + n], k=dotIdx, n=bCol
//         regN[j] = Bs[dotIdx * BN + bCol];
//       }
//       for (int i = 0; i < TM; i++)
//       {
//         for (int j = 0; j < TN; j++)
//         {
//           threadResults[i * TN + j] += regM[i] * regN[j];
//         }
//       }
//     }

//     __syncthreads();
//   }

//   //==========================
//   // Write results back to global memory
//   //==========================
//   for (int i = 0; i < TM; i++)
//   {
//     int cRow = globalRowStart + threadBaseRow + i;
//     for (int j = 0; j < TN; j++)
//     {
//       int cCol = globalColStart + threadBaseCol + j;
//       if (cRow < M && cCol < M)
//       {
//         C[cRow * M + cCol] = threadResults[i * TN + j];
//       }
//     }
//   }
// }

// double *dA, *dB, *dC;

// void square_dgemm(const int M, const double *A0, const double *B0, double *C0)
// {
//   cudaMalloc(&dA, M * M * sizeof(double));
//   cudaMalloc(&dB, M * M * sizeof(double));
//   cudaMalloc(&dC, M * M * sizeof(double));
//   cudaMemcpy(dA, A0, M * M * sizeof(double), cudaMemcpyHostToDevice);
//   cudaMemcpy(dB, B0, M * M * sizeof(double), cudaMemcpyHostToDevice);

//   dim3 numBlocks(CEIL_DIV(M, BM), CEIL_DIV(M, BN));
//   dim3 threadsPerBlock(BN / TN, BM / TM);
//   // For BM=64, TM=4 => BM/TM=16; BN=64, TN=4 => BN/TN=16; so threads=(16,16)=256 threads per block.

//   square_dgemm_kernel_2d_blocktiling<<<numBlocks, threadsPerBlock>>>(M, dA, dB, dC);
//   cudaDeviceSynchronize();
//   cudaMemcpy(C0, dC, M * M * sizeof(double), cudaMemcpyDeviceToHost);

//   cudaFree(dA);
//   cudaFree(dB);
//   cudaFree(dC);
// }

// const char *dgemm_desc = "Optimized DGEMM with 2D blocktiling, transposed shared memory, and vectorized loads.";
