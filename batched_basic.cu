
#include <hip/hip_runtime.h>
#include <stdio.h>
const char *dgemm_desc = "My Basic Batched.";

#define CEIL_DIV(a, b) ((a + b - 1) / b)

// Tile sizes
#define BK 16

// Thread tile sizes
#define TM 4
#define TN 4

// Use __ldg for read-only global memory fetches
// This can sometimes improve performance if the data is read-only
// and may better utilize caches.
__device__ __forceinline__ double ldg_double(const double *ptr) {
#if __CUDA_ARCH__ >= 350
  return __ldg(ptr);
#else
  return *ptr;
#endif
}

// A helper function to load a double2 from global memory safely.
// We'll assume the pointers are suitably aligned since we know BM,BK,BN are multiples of 2.
__device__ __forceinline__ double2 ldg_double2(const double2 *ptr) {
#if __CUDA_ARCH__ >= 350
  return __ldg(ptr);
#else
  return *ptr;
#endif
}

// Compute tile indices with a linear-to-2D mapping for potential improved locality
__device__ inline void compute_tile_indices(int &tile_m, int &tile_n, int m_tiles, int n_tiles) {
  int linear_idx = blockIdx.y * gridDim.x + blockIdx.x;
  tile_m = linear_idx / n_tiles;
  tile_n = linear_idx % n_tiles;
}

//_________________________________________________________
//***********************64*******************************/
#define BM64 64
#define BN64 64

__device__ inline void load_tile_64(
    const double * __restrict__ A, const double * __restrict__ B,
    int M, int globalRowStart, int globalColStart,
    int kt, // which k-tile
    double * __restrict__ As, double * __restrict__ Bs,
    int tRow, int tCol,
    int THREADS_PER_BLOCK_M, int THREADS_PER_BLOCK_N)
{
  // Each tile: A: BMxBK = 64x16=1024 elems, B: BKxBN=16x64=1024 elems
  // blockDim = 16x16 = 256 threads
  // Each thread loads 4 elements from A and 4 from B. We can vectorize these loads using double2.

  int linearThreadIdx = tRow * THREADS_PER_BLOCK_N + tCol;

  int elemsPerThreadA = (BM64 * BK) / (THREADS_PER_BLOCK_M * THREADS_PER_BLOCK_N); // 1024/256=4
  int elemsPerThreadB = (BK * BN64) / (THREADS_PER_BLOCK_M * THREADS_PER_BLOCK_N); // also 4

  // We will load 4 elements from A and B. Since BK=16 is divisible by 2, we can load double2 pairs.
  // Similarly for BN=64 and BM=64.

  // Load A tile
  // The A tile spans rows [globalRowStart .. globalRowStart+BM-1], 
  // and columns [kt*BK .. kt*BK+BK-1].
  // We'll load them linearly and use double2 loads if possible.
  for (int i = 0; i < elemsPerThreadA; i++) {
    int index = linearThreadIdx + i * (THREADS_PER_BLOCK_M * THREADS_PER_BLOCK_N);
    int rowA = index / BK; 
    int colA = index % BK;

    int globalArow = globalRowStart + rowA;
    int globalAcol = kt * BK + colA;

    double valA = 0.0;
    if (globalArow < M && globalAcol < M) {
      valA = ldg_double(&A[globalArow * M + globalAcol]);
    }
    As[rowA * BK + colA] = valA;
  }

  // Load B tile
  // B tile spans rows [kt*BK .. kt*BK+BK-1], cols [globalColStart..globalColStart+BN-1]
  for (int i = 0; i < elemsPerThreadB; i++) {
    int index = linearThreadIdx + i * (THREADS_PER_BLOCK_M * THREADS_PER_BLOCK_N);
    int rowB = index / BN64;
    int colB = index % BN64;

    int globalBrow = kt * BK + rowB;
    int globalBcol = globalColStart + colB;

    double valB = 0.0;
    if (globalBrow < M && globalBcol < M) {
      valB = ldg_double(&B[globalBrow * M + globalBcol]);
    }
    Bs[rowB * BN64 + colB] = valB;
  }
}

__device__ inline void compute_tile_64(
    const double * __restrict__ As, const double * __restrict__ Bs,
    double * __restrict__ threadResults,
    int tRow, int tCol)
{
  // Compute partial product: Ctile += A_tile * B_tile
  // TM=4, TN=4
  #pragma unroll
  for (int dotIdx = 0; dotIdx < BK; ++dotIdx) {
    double regM[TM];
    double regN[TN];

    #pragma unroll
    for (int i = 0; i < TM; i++) {
      int aRow = tRow * TM + i;
      regM[i] = As[aRow * BK + dotIdx];
    }
    #pragma unroll
    for (int j = 0; j < TN; j++) {
      int bCol = tCol * TN + j;
      regN[j] = Bs[dotIdx * BN64 + bCol];
    }
    #pragma unroll
    for (int i = 0; i < TM; i++) {
      double m_val = regM[i];
      #pragma unroll
      for (int j = 0; j < TN; j++) {
        threadResults[i * TN + j] += m_val * regN[j];
      }
    }
  }
}

__global__ void square_dgemm_kernel_2d_blocktiling_64(
    const int M, const int N, const double ** A0, const double ** B0, double ** C0)
{
  // Double buffering
  __shared__ double As[2][BM64 * BK];
  __shared__ double Bs[2][BK * BN64];

  int m_tiles = CEIL_DIV(M, BM64);
  int n_tiles = CEIL_DIV(M, BN64);

  int tile_m, tile_n;
  compute_tile_indices(tile_m, tile_n, m_tiles, n_tiles);

  int batch = blockIdx.z; 

  if (tile_m >= m_tiles || tile_n >= n_tiles || batch >= N)
    return;


    const double *A = A0[batch];
    const double *B = B0[batch];
    double *C = C0[batch];

  const int THREADS_PER_BLOCK_M = BM64 / TM; // 64/4=16
  const int THREADS_PER_BLOCK_N = BN64 / TN; // 64/4=16
  // blockDim should be (16,16)=256 threads

  const int tRow = threadIdx.y; 
  const int tCol = threadIdx.x; 

  const int globalRowStart = tile_m * BM64;
  const int globalColStart = tile_n * BN64;

  double threadResults[TM * TN];
  #pragma unroll
  for (int i = 0; i < TM * TN; i++) {
    threadResults[i] = 0.0;
  }

  int k_tiles = CEIL_DIV(M, BK);

  // Preload the first tile
  int curLoad = 0;
  load_tile_64(A, B, M, globalRowStart, globalColStart, 0,
            As[curLoad], Bs[curLoad], tRow, tCol, THREADS_PER_BLOCK_M, THREADS_PER_BLOCK_N);
  __syncthreads(); // ensure first tile is loaded

  int curCompute = curLoad;

  // Main loop over K
  for (int kt = 1; kt < k_tiles; kt++) {
    int nextLoad = 1 - curLoad;

    // Start loading next tile
    load_tile_64(A, B, M, globalRowStart, globalColStart, kt,
              As[nextLoad], Bs[nextLoad],
              tRow, tCol, THREADS_PER_BLOCK_M, THREADS_PER_BLOCK_N);

    __syncthreads(); // Wait for next tile to be fully loaded

    // Compute on previously loaded tile
    compute_tile_64(As[curCompute], Bs[curCompute], threadResults, tRow, tCol);

    __syncthreads(); // Ensure compute done before reusing shared memory

    // Switch buffers
    curLoad = nextLoad;
    curCompute = curLoad;
  }

  // Compute on the last tile
  compute_tile_64(As[curCompute], Bs[curCompute], threadResults, tRow, tCol);

  // Write results back to C
  #pragma unroll
  for (int i = 0; i < TM; i++) {
    int cRow = globalRowStart + tRow * TM + i;
    #pragma unroll
    for (int j = 0; j < TN; j++) {
      int cCol = globalColStart + tCol * TN + j;
      if (cRow < M && cCol < M) {
        C[cRow * M + cCol] = threadResults[i * TN + j];
      }
    }
  }
}
//********************************************************/
//_________________________________________________________






//_________________________________________________________
//***********************32*******************************/
#define BM32 32
#define BN32 32

__device__ inline void load_tile_32(
    const double * __restrict__ A, const double * __restrict__ B,
    int M, int globalRowStart, int globalColStart,
    int kt, // which k-tile
    double * __restrict__ As, double * __restrict__ Bs,
    int tRow, int tCol,
    int THREADS_PER_BLOCK_M, int THREADS_PER_BLOCK_N)
{
  // Each tile: A: BMxBK = 64x16=1024 elems, B: BKxBN=16x64=1024 elems
  // blockDim = 16x16 = 256 threads
  // Each thread loads 4 elements from A and 4 from B. We can vectorize these loads using double2.

  int linearThreadIdx = tRow * THREADS_PER_BLOCK_N + tCol;

  int elemsPerThreadA = (BM32 * BK) / (THREADS_PER_BLOCK_M * THREADS_PER_BLOCK_N); // 1024/256=4
  int elemsPerThreadB = (BK * BN32) / (THREADS_PER_BLOCK_M * THREADS_PER_BLOCK_N); // also 4

  // We will load 4 elements from A and B. Since BK=16 is divisible by 2, we can load double2 pairs.
  // Similarly for BN=64 and BM=64.

  // Load A tile
  // The A tile spans rows [globalRowStart .. globalRowStart+BM-1], 
  // and columns [kt*BK .. kt*BK+BK-1].
  // We'll load them linearly and use double2 loads if possible.
  for (int i = 0; i < elemsPerThreadA; i++) {
    int index = linearThreadIdx + i * (THREADS_PER_BLOCK_M * THREADS_PER_BLOCK_N);
    int rowA = index / BK; 
    int colA = index % BK;

    int globalArow = globalRowStart + rowA;
    int globalAcol = kt * BK + colA;

    double valA = 0.0;
    if (globalArow < M && globalAcol < M) {
      valA = ldg_double(&A[globalArow * M + globalAcol]);
    }
    As[rowA * BK + colA] = valA;
  }

  // Load B tile
  // B tile spans rows [kt*BK .. kt*BK+BK-1], cols [globalColStart..globalColStart+BN-1]
  for (int i = 0; i < elemsPerThreadB; i++) {
    int index = linearThreadIdx + i * (THREADS_PER_BLOCK_M * THREADS_PER_BLOCK_N);
    int rowB = index / BN32;
    int colB = index % BN32;

    int globalBrow = kt * BK + rowB;
    int globalBcol = globalColStart + colB;

    double valB = 0.0;
    if (globalBrow < M && globalBcol < M) {
      valB = ldg_double(&B[globalBrow * M + globalBcol]);
    }
    Bs[rowB * BN32 + colB] = valB;
  }
}

__device__ inline void compute_tile_32(
    const double * __restrict__ As, const double * __restrict__ Bs,
    double * __restrict__ threadResults,
    int tRow, int tCol)
{
  // Compute partial product: Ctile += A_tile * B_tile
  // TM=4, TN=4
  #pragma unroll
  for (int dotIdx = 0; dotIdx < BK; ++dotIdx) {
    double regM[TM];
    double regN[TN];

    #pragma unroll
    for (int i = 0; i < TM; i++) {
      int aRow = tRow * TM + i;
      regM[i] = As[aRow * BK + dotIdx];
    }
    #pragma unroll
    for (int j = 0; j < TN; j++) {
      int bCol = tCol * TN + j;
      regN[j] = Bs[dotIdx * BN32 + bCol];
    }
    #pragma unroll
    for (int i = 0; i < TM; i++) {
      double m_val = regM[i];
      #pragma unroll
      for (int j = 0; j < TN; j++) {
        threadResults[i * TN + j] += m_val * regN[j];
      }
    }
  }
}

__global__ void square_dgemm_kernel_2d_blocktiling_32(
    const int M, const int N, const double ** A0, const double ** B0, double ** C0)
{
  // Double buffering
  __shared__ double As[2][BM32 * BK];
  __shared__ double Bs[2][BK * BN32];

  int m_tiles = CEIL_DIV(M, BM32);
  int n_tiles = CEIL_DIV(M, BN32);

  int tile_m, tile_n;
  compute_tile_indices(tile_m, tile_n, m_tiles, n_tiles);

  int batch = blockIdx.z; 

  if (tile_m >= m_tiles || tile_n >= n_tiles || batch >= N)
    return;


    const double *A = A0[batch];
    const double *B = B0[batch];
    double *C = C0[batch];

  const int THREADS_PER_BLOCK_M = BM32 / TM; // 64/4=16
  const int THREADS_PER_BLOCK_N = BN32 / TN; // 64/4=16
  // blockDim should be (16,16)=256 threads

  const int tRow = threadIdx.y; 
  const int tCol = threadIdx.x; 

  const int globalRowStart = tile_m * BM32;
  const int globalColStart = tile_n * BN32;

  double threadResults[TM * TN];
  #pragma unroll
  for (int i = 0; i < TM * TN; i++) {
    threadResults[i] = 0.0;
  }

  int k_tiles = CEIL_DIV(M, BK);

  // Preload the first tile
  int curLoad = 0;
  load_tile_32(A, B, M, globalRowStart, globalColStart, 0,
            As[curLoad], Bs[curLoad], tRow, tCol, THREADS_PER_BLOCK_M, THREADS_PER_BLOCK_N);
  __syncthreads(); // ensure first tile is loaded

  int curCompute = curLoad;

  // Main loop over K
  for (int kt = 1; kt < k_tiles; kt++) {
    int nextLoad = 1 - curLoad;

    // Start loading next tile
    load_tile_32(A, B, M, globalRowStart, globalColStart, kt,
              As[nextLoad], Bs[nextLoad],
              tRow, tCol, THREADS_PER_BLOCK_M, THREADS_PER_BLOCK_N);

    __syncthreads(); // Wait for next tile to be fully loaded

    // Compute on previously loaded tile
    compute_tile_32(As[curCompute], Bs[curCompute], threadResults, tRow, tCol);

    __syncthreads(); // Ensure compute done before reusing shared memory

    // Switch buffers
    curLoad = nextLoad;
    curCompute = curLoad;
  }

  // Compute on the last tile
  compute_tile_32(As[curCompute], Bs[curCompute], threadResults, tRow, tCol);

  // Write results back to C
  #pragma unroll
  for (int i = 0; i < TM; i++) {
    int cRow = globalRowStart + tRow * TM + i;
    #pragma unroll
    for (int j = 0; j < TN; j++) {
      int cCol = globalColStart + tCol * TN + j;
      if (cRow < M && cCol < M) {
        C[cRow * M + cCol] = threadResults[i * TN + j];
      }
    }
  }
}
//********************************************************/
//_________________________________________________________

//_________________________________________________________
//***********************16*******************************/
#define BM16 16
#define BN16 16

__device__ inline void load_tile_16(
    const double * __restrict__ A, const double * __restrict__ B,
    int M, int globalRowStart, int globalColStart,
    int kt, // which k-tile
    double * __restrict__ As, double * __restrict__ Bs,
    int tRow, int tCol,
    int THREADS_PER_BLOCK_M, int THREADS_PER_BLOCK_N)
{
  // Each tile: A: BMxBK = 64x16=1024 elems, B: BKxBN=16x64=1024 elems
  // blockDim = 16x16 = 256 threads
  // Each thread loads 4 elements from A and 4 from B. We can vectorize these loads using double2.

  int linearThreadIdx = tRow * THREADS_PER_BLOCK_N + tCol;

  int elemsPerThreadA = (BM16 * BK) / (THREADS_PER_BLOCK_M * THREADS_PER_BLOCK_N); // 1024/256=4
  int elemsPerThreadB = (BK * BN16) / (THREADS_PER_BLOCK_M * THREADS_PER_BLOCK_N); // also 4

  // We will load 4 elements from A and B. Since BK=16 is divisible by 2, we can load double2 pairs.
  // Similarly for BN=64 and BM=64.

  // Load A tile
  // The A tile spans rows [globalRowStart .. globalRowStart+BM-1], 
  // and columns [kt*BK .. kt*BK+BK-1].
  // We'll load them linearly and use double2 loads if possible.
  for (int i = 0; i < elemsPerThreadA; i++) {
    int index = linearThreadIdx + i * (THREADS_PER_BLOCK_M * THREADS_PER_BLOCK_N);
    int rowA = index / BK; 
    int colA = index % BK;

    int globalArow = globalRowStart + rowA;
    int globalAcol = kt * BK + colA;

    double valA = 0.0;
    if (globalArow < M && globalAcol < M) {
      valA = ldg_double(&A[globalArow * M + globalAcol]);
    }
    As[rowA * BK + colA] = valA;
  }

  // Load B tile
  // B tile spans rows [kt*BK .. kt*BK+BK-1], cols [globalColStart..globalColStart+BN-1]
  for (int i = 0; i < elemsPerThreadB; i++) {
    int index = linearThreadIdx + i * (THREADS_PER_BLOCK_M * THREADS_PER_BLOCK_N);
    int rowB = index / BN16;
    int colB = index % BN16;

    int globalBrow = kt * BK + rowB;
    int globalBcol = globalColStart + colB;

    double valB = 0.0;
    if (globalBrow < M && globalBcol < M) {
      valB = ldg_double(&B[globalBrow * M + globalBcol]);
    }
    Bs[rowB * BN16 + colB] = valB;
  }
}

__device__ inline void compute_tile_16(
    const double * __restrict__ As, const double * __restrict__ Bs,
    double * __restrict__ threadResults,
    int tRow, int tCol)
{
  // Compute partial product: Ctile += A_tile * B_tile
  // TM=4, TN=4
  #pragma unroll
  for (int dotIdx = 0; dotIdx < BK; ++dotIdx) {
    double regM[TM];
    double regN[TN];

    #pragma unroll
    for (int i = 0; i < TM; i++) {
      int aRow = tRow * TM + i;
      regM[i] = As[aRow * BK + dotIdx];
    }
    #pragma unroll
    for (int j = 0; j < TN; j++) {
      int bCol = tCol * TN + j;
      regN[j] = Bs[dotIdx * BN16 + bCol];
    }
    #pragma unroll
    for (int i = 0; i < TM; i++) {
      double m_val = regM[i];
      #pragma unroll
      for (int j = 0; j < TN; j++) {
        threadResults[i * TN + j] += m_val * regN[j];
      }
    }
  }
}

__global__ void square_dgemm_kernel_2d_blocktiling_16(
    const int M, const int N, const double ** A0, const double ** B0, double ** C0)
{
  // Double buffering
  __shared__ double As[2][BM16 * BK];
  __shared__ double Bs[2][BK * BN16];

  int m_tiles = CEIL_DIV(M, BM16);
  int n_tiles = CEIL_DIV(M, BN16);

  int tile_m, tile_n;
  compute_tile_indices(tile_m, tile_n, m_tiles, n_tiles);

  int batch = blockIdx.z; 

  if (tile_m >= m_tiles || tile_n >= n_tiles || batch >= N)
    return;


    const double *A = A0[batch];
    const double *B = B0[batch];
    double *C = C0[batch];

  const int THREADS_PER_BLOCK_M = BM16 / TM; // 64/4=16
  const int THREADS_PER_BLOCK_N = BN16 / TN; // 64/4=16
  // blockDim should be (16,16)=256 threads

  const int tRow = threadIdx.y; 
  const int tCol = threadIdx.x; 

  const int globalRowStart = tile_m * BM16;
  const int globalColStart = tile_n * BN16;

  double threadResults[TM * TN];
  #pragma unroll
  for (int i = 0; i < TM * TN; i++) {
    threadResults[i] = 0.0;
  }

  int k_tiles = CEIL_DIV(M, BK);

  // Preload the first tile
  int curLoad = 0;
  load_tile_16(A, B, M, globalRowStart, globalColStart, 0,
            As[curLoad], Bs[curLoad], tRow, tCol, THREADS_PER_BLOCK_M, THREADS_PER_BLOCK_N);
  __syncthreads(); // ensure first tile is loaded

  int curCompute = curLoad;

  // Main loop over K
  for (int kt = 1; kt < k_tiles; kt++) {
    int nextLoad = 1 - curLoad;

    // Start loading next tile
    load_tile_16(A, B, M, globalRowStart, globalColStart, kt,
              As[nextLoad], Bs[nextLoad],
              tRow, tCol, THREADS_PER_BLOCK_M, THREADS_PER_BLOCK_N);

    __syncthreads(); // Wait for next tile to be fully loaded

    // Compute on previously loaded tile
    compute_tile_16(As[curCompute], Bs[curCompute], threadResults, tRow, tCol);

    __syncthreads(); // Ensure compute done before reusing shared memory

    // Switch buffers
    curLoad = nextLoad;
    curCompute = curLoad;
  }

  // Compute on the last tile
  compute_tile_16(As[curCompute], Bs[curCompute], threadResults, tRow, tCol);

  // Write results back to C
  #pragma unroll
  for (int i = 0; i < TM; i++) {
    int cRow = globalRowStart + tRow * TM + i;
    #pragma unroll
    for (int j = 0; j < TN; j++) {
      int cCol = globalColStart + tCol * TN + j;
      if (cRow < M && cCol < M) {
        C[cRow * M + cCol] = threadResults[i * TN + j];
      }
    }
  }
}
//********************************************************/
//_________________________________________________________



// Host function that launches the kernel
// We assume that A, B, C are device arrays of pointers to device matrices.
void batched_gemm_kernel
( 
    const int M, 
    const int N, 
    const double **A0, 
    const double **B0, 
    double **C0
)
{


    dim3 blockDim;
    dim3 gridDim;

    // dim3 blockDim(16, 16);
    // dim3 gridDim((M + blockDim.x - 1) / blockDim.x,
    //              (M + blockDim.y - 1) / blockDim.y,
    //              N);

    size_t matrix_size = M * M * sizeof(double);

    // Allocate device memory for matrices
    double *d_A_data, *d_B_data, *d_C_data;
    hipMalloc((void **)&d_A_data, N * matrix_size);
    hipMalloc((void **)&d_B_data, N * matrix_size);
    hipMalloc((void **)&d_C_data, N * matrix_size);

    const double **d_A_array, **d_B_array;
          double **d_C_array;
    hipMalloc((void **)&d_A_array, N * sizeof(double *));
    hipMalloc((void **)&d_B_array, N * sizeof(double *));
    hipMalloc((void **)&d_C_array, N * sizeof(double *));

    // Allocate host memory for device pointers
    double **h_A_list = (double **)malloc(N * sizeof(double *));
    double **h_B_list = (double **)malloc(N * sizeof(double *));
    double **h_C_list = (double **)malloc(N * sizeof(double *));

    // Copy input matrices to device memory
    for (int i = 0; i < N; i++) {
        hipMemcpy(d_A_data + i * M * M, A0[i], matrix_size, hipMemcpyHostToDevice);
        hipMemcpy(d_B_data + i * M * M, B0[i], matrix_size, hipMemcpyHostToDevice);
    }

    // Prepare device pointers for the batch
    for (int i = 0; i < N; i++) {
        h_A_list[i] = d_A_data + i * M * M;
        h_B_list[i] = d_B_data + i * M * M;
        h_C_list[i] = d_C_data + i * M * M;
    }

    // Copy the array of device pointers to the GPU
    hipMemcpy(d_A_array, h_A_list, N * sizeof(double *), hipMemcpyHostToDevice);
    hipMemcpy(d_B_array, h_B_list, N * sizeof(double *), hipMemcpyHostToDevice);
    hipMemcpy(d_C_array, h_C_list, N * sizeof(double *), hipMemcpyHostToDevice);

    // Launch the kernel
    
    // Decide which kernel to launch based on M
    if (M > 94){
        // 64x64 Tile Configuration
        blockDim = dim3(16, 16);
        gridDim = dim3(CEIL_DIV(M, BM64), CEIL_DIV(M, BN64), N);

        square_dgemm_kernel_2d_blocktiling_64<<<gridDim, blockDim>>>(M, N, d_A_array, d_B_array, d_C_array);
        // printf("Launched 64x64 tile kernel.\n");
    } 
    else if(M > 40){
        // 32x32 Tile Configuration
        blockDim = dim3(8, 8);
        gridDim = dim3(CEIL_DIV(M, BM32), CEIL_DIV(M, BN32), N);

        square_dgemm_kernel_2d_blocktiling_32<<<gridDim, blockDim>>>(M, N, d_A_array, d_B_array, d_C_array);
        // printf("Launched 32x32 tile kernel.\n");
    } 
    else{
        // 16x16 Tile Configuration
        blockDim = dim3(4, 4);
        gridDim = dim3(CEIL_DIV(M, BM16), CEIL_DIV(M, BN16), N);

        square_dgemm_kernel_2d_blocktiling_16<<<gridDim, blockDim>>>(M, N, d_A_array, d_B_array, d_C_array);
        // printf("Launched 16x16 tile kernel.\n");
    }

    hipDeviceSynchronize();

    // Copy the result back to the host
    for (int i = 0; i < N; i++) {
        hipMemcpy(C0[i], d_C_data + i * M * M, matrix_size, hipMemcpyDeviceToHost);
    }

    // Free device memory
    hipFree(d_A_data);
    hipFree(d_B_data);
    hipFree(d_C_data);
    hipFree(d_A_array);
    hipFree(d_B_array);
    hipFree(d_C_array);

    // Free host memory
    free(h_A_list);
    free(h_B_list);
    free(h_C_list);
}
